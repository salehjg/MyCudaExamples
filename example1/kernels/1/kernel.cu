
#include <hip/hip_runtime.h>
//
// Created by saleh on 9/14/23.
//

__global__
void VecAdd(const float *pIn1, const float *pIn2, float *pOut1) {
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    pOut1[idx] = pIn1[idx] + pIn2[idx];
}

// This function cannot be a template.
// Since different compilers are used to compile *.cu and *.cpp files.
// But the kernel itself could be templated and called in this function's body.
void LaunchVecAdd(unsigned blockSize, size_t len, const float *i1, const float *i2, float *i3) {
    VecAdd<<<(len-1)/blockSize +1, blockSize>>>(i1, i2, i3);
}
